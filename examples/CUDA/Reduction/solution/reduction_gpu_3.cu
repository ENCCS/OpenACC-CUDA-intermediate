
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <time.h>

#define BLOCK_SIZE 256

__device__ __forceinline__ float getValue(const float* data, int index, int numElements)
{
    if(index < numElements)
    {
        return data[index];
    }
    else
    {
        return 0.0f;
    }
}

__global__ void reduce_kernel(const float* data, float* result, int numElements)
{
    extern __shared__ float s_data[];

    int s_i = threadIdx.x;
    int d_i = threadIdx.x + blockIdx.x*blockDim.x;
    s_data[s_i] = getValue(data, d_i, numElements);
    
    for (int offset = 1; offset < blockDim.x; offset *= 2)
    {
        __syncthreads();
        if (s_i % (2*offset) == 0)
        {
            s_data[s_i] += s_data[s_i + offset];
        }
    }

    if (s_i == 0)
    {
        result[blockIdx.x] = s_data[0];
    }
}

int main(int argc, char* argv[])
{

    int numElements = (argc > 1) ? atoi(argv[1]) : 100000000;

    printf("Reducing over %d values.\n", numElements);

    float* h_data   = (float*)calloc(numElements, sizeof(float));

    srand(1214134);
    for (int i = 0; i < numElements; i++)
    {
        h_data[i] = float(rand())/float(RAND_MAX + 1.0);
    }

    float h_result = 0.0;

    float* d_data;
    hipMalloc((void**)&d_data, numElements*sizeof(float));
    hipMemcpy(d_data, h_data, numElements*sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = BLOCK_SIZE;
    int numBlocks = numElements/BLOCK_SIZE + 1;

    float* d_result1;
    float* d_result2;
    hipMalloc((void**)&d_result1, numBlocks*sizeof(float));
    hipMalloc((void**)&d_result2, numBlocks*sizeof(float));

    // Timing
    clock_t start = clock();

    // Main loop
    reduce_kernel<<<numBlocks, threadsPerBlock, threadsPerBlock*sizeof(float)>>>(d_data, d_result1, numElements);
    for (int numElementsCurrent = numBlocks; numElementsCurrent > 1; )
    {
        int numBlocksCurrent = numElementsCurrent/BLOCK_SIZE + 1;
        reduce_kernel<<<numBlocksCurrent, threadsPerBlock, threadsPerBlock*sizeof(float)>>>(d_result1, d_result2, numElementsCurrent);
        numElementsCurrent = numBlocksCurrent;
        std::swap(d_result1, d_result2);
    }

    hipMemcpy(&h_result, d_result1, 1*sizeof(float), hipMemcpyDeviceToHost);

    // Timing
    clock_t finish = clock();

    printf("The result is: %f\n", h_result);

    printf("It took %f seconds\n", (double)(finish - start) / CLOCKS_PER_SEC);

    // Release the memory
    free(h_data);
    hipFree(d_data);
    hipFree(d_result1);
    hipFree(d_result2);
    
    return 0;
}