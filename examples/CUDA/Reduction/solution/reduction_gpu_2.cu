
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <time.h>

#define BLOCK_SIZE 256

__device__ __forceinline__ float getValue(const float* data, int index, int numElements)
{
    if(index < numElements)
    {
        return data[index];
    }
    else
    {
        return 0.0f;
    }
}

__global__ void reduce_kernel(const float* data, float* result, int numElements)
{
    int d_i = threadIdx.x + blockIdx.x*blockDim.x;
    
    result[d_i] = getValue(data, 2*d_i, numElements) + getValue(data, 2*d_i + 1, numElements);

    if (d_i == 0 && numElements % 2 != 0)
    {
        result[d_i] += data[numElements-1];
    }
}

int main(int argc, char* argv[])
{

    int numElements = (argc > 1) ? atoi(argv[1]) : 100000000;

    printf("Reducing over %d values.\n", numElements);

    float* h_data   = (float*)calloc(numElements, sizeof(float));

    srand(1214134);
    for (int i = 0; i < numElements; i++)
    {
        h_data[i] = float(rand())/float(RAND_MAX + 1.0);
    }

    float h_result = 0.0;

    float* d_data;
    hipMalloc((void**)&d_data, numElements*sizeof(float));
    hipMemcpy(d_data, h_data, numElements*sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = BLOCK_SIZE;
    int numBlocks = numElements/2/BLOCK_SIZE + 1;

    float* d_result1;
    float* d_result2;
    hipMalloc((void**)&d_result1, numElements*sizeof(float));
    hipMalloc((void**)&d_result2, numElements*sizeof(float));

    // Timing
    clock_t start = clock();

    // Main loop
    reduce_kernel<<<numBlocks, threadsPerBlock>>>(d_data, d_result1, numElements);
    for (int numElementsCurrent = numElements/2; numElementsCurrent > 1; numElementsCurrent = numElementsCurrent/2)
    {
        int numBlocksCurrent = numElementsCurrent/2/BLOCK_SIZE + 1;
        reduce_kernel<<<numBlocksCurrent, threadsPerBlock>>>(d_result1, d_result2, numElementsCurrent);
        std::swap(d_result1, d_result2);
    }

    hipMemcpy(&h_result, d_result1, 1*sizeof(float), hipMemcpyDeviceToHost);

    // Timing
    clock_t finish = clock();

    printf("The result is: %f\n", h_result);

    printf("It took %f seconds\n", (double)(finish - start) / CLOCKS_PER_SEC);

    // Release the memory
    free(h_data);
    hipFree(d_data);
    hipFree(d_result1);
    hipFree(d_result2);
    
    return 0;
}