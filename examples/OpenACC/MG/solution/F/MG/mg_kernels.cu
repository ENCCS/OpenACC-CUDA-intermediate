#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

//----------------------------------------------------------------
// Check the last CUDA error
void checkCUDAError(const char *msg){
  hipError_t err = hipGetLastError();
  if(hipSuccess!=err){
    fprintf(stderr,"CUDA Error: %s: %s.\n",msg,hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
//----------------------------------------------------------------

__global__ void resid_kernel(double *u, double *v, double *r, 
			     int n1, int n2, int n3, 
			     double *a)
{
  // Number of this thread
  int tidx = blockIdx.x*blockDim.x+threadIdx.x;
  // Total number of threads
  int tnum = gridDim.x*blockDim.x;

  int n12 = n1*n2;
    
  // Loop over all the iterations needed
  for(int i123=tidx;i123 < n12*n3;i123+=tnum){

    int i3 = i123/n12;
    int i12 = i123 - i3*n12;
    int i2 = i12/n1;
    int i1 = i12 - i2*n1;
    
    // Only do calculations if we are in the bulk (not the halo)
    if (i1 > 0 && i1 < n1-1 &&
	i2 > 0 && i2 < n2-1 &&
	i3 > 0 && i3 < n3-1)
      {
	r[i123] = v[i123]
	  - a[0] * u[i123]
	  - a[2] * ( 
                 u[i123-n1-n12] + u[i123+n1-n12]
                 + u[i123-n1+n12] + u[i123+n1+n12] +
                 u[i123-1-n1] + u[i123-1+n1]
                 + u[i123-1-n12] + u[i123-1+n12] +
                 u[i123+1-n1] + u[i123+1+n1]
                 + u[i123+1-n12] + u[i123+1+n12] )
	  - a[3] * ( 
                 u[i123-1-n1-n12] + u[i123-1+n1-n12]
                 + u[i123-1-n1+n12] + u[i123-1+n1+n12] +
                 u[i123+1-n1-n12] + u[i123+1+n1-n12]
                 + u[i123+1-n1+n12] + u[i123+1+n1+n12] );
      }
  }
}

extern "C" void resid_cuda_(double *u, double *v, double *r, 
			     int *n1, int *n2, int *n3, 
			     double *a)
{
    // Synchronise threads to make sure OpenACC kernels have finished
    hipDeviceSynchronize();

    // Kernel launch parameters
    int threads_n = NTHREADS;
    int n123 = (*n1)*(*n2)*(*n3);
    int blocks_n = (n123+threads_n-1)/threads_n;
    // Make sure there aren't too many blocks.
    if (blocks_n > 65535) blocks_n = 65535;

    // Launch the GPU kernel
    resid_kernel<<<blocks_n,threads_n>>>(u, v, r, 
					 *n1, *n2, *n3, 
					 a);

    // Synchronise threads before more OpenACC kernels started
    hipDeviceSynchronize();
    // Check the error (commented out)
    //checkCUDAError("kernel");
}
